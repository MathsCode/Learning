#include "kernel.h"
#include <stdio.h>
#define n_streams 4
int main()
{
    hipStream_t *streams = (hipStream_t *)malloc(n_streams * sizeof(hipStream_t)); 
    for (int i = 0 ; i < n_streams; i++)  hipStreamCreate(&streams[i]); 
    hipEvent_t start, stop; 
    hipEventCreate(&start); 
    hipEventCreate(&stop);
    dim3 block(1); 
    dim3 grid(1); 
    int num = 128;
    hipEventRecord(start);
    for (int i = 0; i < n_streams; i++) 
    { 
        kernel_1<<<grid, block, 0, streams[i]>>>(num); 
        kernel_2<<<grid, block, 0, streams[i]>>>(num);
        kernel_3<<<grid, block, 0, streams[i]>>>(num); 
        kernel_4<<<grid, block, 0, streams[i]>>>(num); 
    }
    hipEventRecord(stop);
    float elapsed_time =0.0;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed_time, start, stop);
    printf("%lf",elapsed_time);
}